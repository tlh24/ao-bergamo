#include "hip/hip_runtime.h"
// process an image using CUDA. 
#include <iostream>
#include <stdio.h>
#define MAX_THREADS 256
#define SIZE (2048*2048)

unsigned char* d_image; 
float* d_results; 
int g_imagew; 
int g_imageh; 

void cu_setup(int width, int height, int nresults){
	hipMalloc(&d_image, width*height); 
	hipMalloc(&d_results, nresults); 
	if(d_image == 0 || d_results == 0){
		std::cout << "could not allocate CUDA memory" << endl; 
		imagew = imageh = 0
	} else {
		g_imagew = width; 
		g_imageh = height; 
	}
}

__global__ void sumall_kernel(unsigned char *dat, float *d_result){
    __shared__ float cache[MAX_THREADS];

    int cacheIdx = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cache[cacheIdx] = i >= SIZE ? 0. : d_vector[i];
    __syncthreads();

    if (i >= SIZE)
        return;

    int padding = blockDim.x/2;
    while (padding != 0)
    {
        if (cacheIdx < padding)
            cache[cacheIdx] += cache[cacheIdx + padding];

        __syncthreads();
        padding /= 2;
    }

    if (cacheIdx == 0)
        atomicAdd(&d_result[0], cache[0]);
}

void cu_process(unsigned char* indata){
	
}
